/*
 testeando
 */

#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

const static int threadsPerBlock = 1024;
const static int blocksPerGrid = 10;
const static int M = 3;
/////////

__global__ void dot_pdt(float* d_vtr_a, float* d_vtr_b, const int M,
		float* d_vtr_o);

int parseCLI(const int argc, char**argv, char *filename_0, char *filename_1);
void read_mtx(const char *in_filename, float *mtx, const unsigned int M,
		const unsigned int N);

/////////

int main(int argc, char** argv) {

	// CPU
	char in_filename_1[20], in_filename_2[20];
	float *vtr_a, *vtr_b, *vtr_out;

	vtr_a = (float*) malloc(M * sizeof(float));
	vtr_b = (float*) malloc(M * sizeof(float));
	vtr_out = (float*) malloc(blocksPerGrid * sizeof(float));

//	parseCLI(argc, argv, in_filename_1, in_filename_2);
//	read_mtx(in_filename_1, vtr_a, M, 1);
//	read_mtx(in_filename_2, vtr_b, M, 1);
	vtr_a[0] = 1.;
	vtr_a[1] = 1.;
	vtr_a[2] = 1.;
	vtr_b[0] = 1.;
	vtr_b[1] = 1.;
	vtr_b[2] = 1.;

//
	printf("Vtor a:\n");
	for (size_t i = 0; i < M; i++) {
		printf("%f\t", vtr_a[i]);
	}
	printf("\nVtor b:\n");
	for (size_t i = 0; i < M; i++) {
		printf("%f\t", vtr_b[i]);
	}
	printf("\n");
//

// GPU
	dim3 dimBlock(threadsPerBlock, 1, 1);
	dim3 dimGrid(blocksPerGrid, 1, 1);
	float *d_vtr_a, *d_vtr_b, *d_vtr_out;
	hipMalloc((void**) &d_vtr_a, M * sizeof(float));
	hipMalloc((void**) &d_vtr_b, M * sizeof(float));
	hipMalloc((void**) &d_vtr_out, blocksPerGrid * sizeof(float));
	hipMemcpy(d_vtr_a, vtr_a, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_vtr_b, vtr_b, M * sizeof(float), hipMemcpyHostToDevice);

	dot_pdt<<<dimGrid, dimBlock>>>(d_vtr_a, d_vtr_b, M, d_vtr_out);

	hipMemcpy(vtr_out, d_vtr_out, M * sizeof(float), hipMemcpyDeviceToHost);

	float result = 0.;
	for (size_t i = 0; i < M; ++i) {
		printf("vtr_out:  %f\n", vtr_out[i]);
		result += vtr_out[i];
	}
	printf("dot pdt:  %f\n", result);

	return 0;
}

///////// Kernels

__global__ void dot_pdt(float* d_vtr_a, float* d_vtr_b, const int M,
		float* d_vtr_o) {

	int ti = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ float tmp_sum[threadsPerBlock];
	tmp_sum[threadIdx.x] = 0;

	float tmp = 0;
	while (ti < M) {
		tmp += d_vtr_a[ti] * d_vtr_b[ti];
		ti += blockDim.x * gridDim.x;
	}
	tmp_sum[threadIdx.x] = tmp;
	__syncthreads();

	// Ahora tengo q reducir tmp_sum
	size_t idx = blockDim.x / 2;
	while (idx != 0) {
		if (threadIdx.x < idx) {
			tmp_sum[threadIdx.x] += tmp_sum[threadIdx.x + idx];
		}
		__syncthreads();
		idx /= 2;
	}
	if (threadIdx.x == 0) {
		d_vtr_o[blockIdx.x] = tmp_sum[0];
	}

	return;
}

///////// Functions

int parseCLI(const int argc, char**argv, char *filename_0, char *filename_1) {

	bool m_flag, v_flag;
	char c;

	while ((c = getopt(argc, argv, "m:v:")) != -1) {
		switch (c) {
		case 'm':
			if (sizeof(optarg) > 20) {
				fprintf(stderr, "Filename too big.\n");
				return 1;
			}

			memset(filename_0, '\0', sizeof(optarg));
			strcpy(filename_0, optarg);
			m_flag = 1;
			break;
		case 'v':
			if (sizeof(optarg) > 20) {
				fprintf(stderr, "Filename too big.\n");
				return 1;
			}

			memset(filename_1, '\0', sizeof(optarg));
			strcpy(filename_1, optarg);
			filename_1 = optarg;
			v_flag = 1;
			break;
		case '?':
			if (optopt == 'i' || optopt == 'o')
				fprintf(stderr, "Options require an argument.\n");
			return 1;
		default:
			fprintf(stderr, "Mal\n");
			return 1;
		}
	}
	if ((m_flag && v_flag) != 1) {
		fprintf(stderr, "Mal\n");
		return 1;
	}

	return 0;
}

void read_mtx(const char *in_filename, float *mtx, const unsigned int M,
		const unsigned int N) {

	FILE *in_file;
	in_file = fopen(in_filename, "r");

	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			fscanf(in_file, "%f\t", &mtx[j + (i * N)]);
		}
	}

	return;
}
